#include "hip/hip_runtime.h"
// ***************************************************************
// Copyright (c) 2021 Jittor. All Rights Reserved.
// This file is subject to the terms and conditions defined in
// file 'LICENSE.txt', which is part of this source code package.
// ***************************************************************
#include "misc/nan_checker.h"
#include "misc/cuda_flags.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <cassert>

namespace jittor {


#ifdef HAS_CUDA
__global__ void _check_nan_float32(float32* __restrict__ ptr, int64 num) {
    int64 i = threadIdx.x + blockIdx.x * (int64)blockDim.x;
    if (i<num) {
        if (::isnan(ptr[i]) || ::isinf(ptr[i]))
            __trap();
    }
}


__global__ void _check_nan_float64(float64* __restrict__ ptr, int64 num) {
    int64 i = threadIdx.x + blockIdx.x * (int64)blockDim.x;
    if (i<num) {
        if (::isnan(ptr[i]) || ::isinf(ptr[i]))
            __trap();
    }
}

void check_nan_float64(float64* ptr, int64 num) {
    int block_num = std::max((int64)1, (num-1)/1024+1);
    int thread_num = std::min((int64)1024, num);
    _check_nan_float64<<<block_num, thread_num>>>(ptr, num);
}

void check_nan_float32(float32* ptr, int64 num) {
    int block_num = std::max((int64)1, (num-1)/1024+1);
    int thread_num = std::min((int64)1024, num);
    _check_nan_float32<<<block_num, thread_num>>>(ptr, num);
}

#endif

}