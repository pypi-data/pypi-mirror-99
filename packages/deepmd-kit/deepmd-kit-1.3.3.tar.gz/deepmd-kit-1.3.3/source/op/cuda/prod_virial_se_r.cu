#include "hip/hip_runtime.h"
#include "DeviceFunctor.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

template<typename FPTYPE>
__global__ void deriv_wrt_neighbors_se_r(FPTYPE * virial, 
                        FPTYPE * atom_virial,
                        const FPTYPE * net_deriv,
                        const FPTYPE * in_deriv,
                        const FPTYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nnei,
                        const int ndescrpt) 
{
    // idx -> nloc
    // idy -> nnei
    // idz = dd0 * 3 + dd1
    // dd0 = idz / 3
    // dd1 = idz % 3
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;

    if (idx >= nloc) {
        return;
    }
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    // atomicAdd(virial + idz, net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz % 3]);
    atomicAdd(atom_virial + j_idx * 9 + idz, net_deriv[idx * ndescrpt + idy] * rij[idx * nnei * 3 + idy * 3 + idz % 3] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz / 3]);
}

template <typename FPTYPE>
void ProdVirialSeRGPUExecuteFunctor<FPTYPE>::operator()(FPTYPE * virial, 
                        FPTYPE * atom_virial,
                        const FPTYPE * net_deriv,
                        const FPTYPE * in_deriv,
                        const FPTYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nall,
                        const int nnei,
                        const int ndescrpt)
{
    cudaErrcheck(hipMemset(virial, 0.0, sizeof(FPTYPE) * 9));
    cudaErrcheck(hipMemset(atom_virial, 0.0, sizeof(FPTYPE) * 9 * nall));

    const int LEN = 64;
    int nblock = (nloc + LEN -1) / LEN;
    dim3 block_grid(nblock, nnei);
    dim3 thread_grid(LEN, 9);
    // compute virial of a frame
    deriv_wrt_neighbors_se_r<<<block_grid, thread_grid>>>(
                        virial, 
                        atom_virial, 
                        net_deriv, 
                        in_deriv,
                        rij,
                        nlist,
                        nloc,
                        nnei,
                        ndescrpt
    );
}

template struct ProdVirialSeRGPUExecuteFunctor<float>;
template struct ProdVirialSeRGPUExecuteFunctor<double>;