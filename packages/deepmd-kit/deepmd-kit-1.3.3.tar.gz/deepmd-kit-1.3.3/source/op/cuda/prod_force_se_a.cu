#include "hip/hip_runtime.h"
#include "DeviceFunctor.h"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

template<typename FPTYPE>
__global__ void deriv_wrt_center_atom_se_a(FPTYPE * force, 
                        const FPTYPE * net_deriv,
                        const FPTYPE * in_deriv,
                        const int ndescrpt)
{
    const unsigned int idx = blockIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idz = threadIdx.x;

    if (idy >= ndescrpt) {return;}
    
    atomicAdd(force + idx * 3 + idz, -1.0 * net_deriv[idx * ndescrpt + idy] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz]);
}

template<typename FPTYPE>
__global__ void deriv_wrt_neighbors_se_a(FPTYPE * force, 
                        const FPTYPE * net_deriv,
                        const FPTYPE * in_deriv,
                        const int * nlist,
                        const int nloc,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift)
{  
    // idy -> nnei
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;
    const unsigned int idw = threadIdx.z;
    
    if (idx >= nloc) {
        return;
    }
    // deriv wrt neighbors
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    atomicAdd(force + j_idx * 3 + idz, net_deriv[idx * ndescrpt + idy * 4 + idw] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz]);
}

template <typename FPTYPE>
void ProdForceSeAGPUExecuteFunctor<FPTYPE>::operator()(FPTYPE * force, 
                        const FPTYPE * net_deriv,
                        const FPTYPE * in_deriv,
                        const int * nlist,
                        const int nloc,
                        const int nall,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift)
{   
    // std::cout << "I'm here!" << std::endl;
    cudaErrcheck(hipMemset(force, 0.0, sizeof(FPTYPE) * nall * 3));
    const int LEN1 = 256;
    const int nblock1 = (ndescrpt + LEN1 -1) / LEN1;
    dim3 grid(nloc, nblock1);
    dim3 thread(3, LEN1);
    deriv_wrt_center_atom_se_a<<<grid, thread>>>(force, net_deriv, in_deriv, ndescrpt);
    
    const int LEN = 64;
    int nblock = (nloc + LEN -1) / LEN;
    dim3 block_grid(nblock, nnei);
    dim3 thread_grid(LEN, 3, 4);
    deriv_wrt_neighbors_se_a<<<block_grid, thread_grid>>>(force, net_deriv, in_deriv, nlist, nloc, nnei, ndescrpt, n_a_sel, n_a_shift);
}

template struct ProdForceSeAGPUExecuteFunctor<float>;
template struct ProdForceSeAGPUExecuteFunctor<double>;